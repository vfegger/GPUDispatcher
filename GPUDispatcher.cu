#include "GPUDispatcher.cuh"

#define ERRORLOG

#ifdef ERRORLOG
#define RAISE_CUDA_ERROR(cudaStatus,errorStatus) \
    if(*errorStatus == 0){ \
        *errorStatus = (int)hipblasStatus_t; \
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) { \
            std::cout << "Error Cuda #" << cudaStatus << " : Cannot proceed with the implementation\n"; \
        } \
    }
#define RAISE_CUBLAS_ERROR(hipblasStatus_t,errorStatus) \
    if(*errorStatus == 0){ \
        *errorStatus = (int)hipblasStatus_t; \
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) { \
            std::cout << "Error Cublas #" << hipblasStatus_t << " : Cannot proceed with the implementation\n"; \
        } \
    }
#else
#define RAISE_CUDA_ERROR(cudaStatus,errorStatus)
#define RAISE_CUBLAS_ERROR(hipblasStatus_t,errorStatus)
#endif

GPUController::GPUController(int sizeElementsX_in, int sizeElementsY_in, int sizeElementsZ_in, int sizeType_in, int* errorStatus) {
    sizeElementX = sizeElementsX_in;
    sizeElementY = sizeElementsY_in;
    sizeElementZ = sizeElementsZ_in;
    sizeType = sizeType_in;

    sizeStream = 0;
    pointer_CPU = NULL;
    pointer_GPU = NULL;
    pointer_Stream_CPU_input = NULL;
    pointer_Stream_GPU_input = NULL;
    pointer_Stream_CPU_result = NULL;
    pointer_Stream_GPU_result = NULL;
    cudaStatus = hipSuccess;
    hipblasStatus_t = HIPBLAS_STATUS_SUCCESS;
    
    cudaStatus = hipSetDevice(0);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    cudaStatus = hipDeviceReset();
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);

    cudaStatus = hipStreamCreate(&cudaStream);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    RAISE_CUBLAS_ERROR(hipblasStatus_t, errorStatus);
    hipblasStatus_t = hipblasSetStream(cublasHandle, cudaStream);
    RAISE_CUBLAS_ERROR(hipblasStatus_t, errorStatus);

    alpha = new double(1.0);
    beta = new double(0.0);
}

void GPUController::InitializeGlobal(int* errorStatus) {
    cudaStatus = hipHostMalloc(&pointer_CPU, sizeElementX * sizeElementY * sizeType);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    cudaStatus = hipMalloc(&pointer_GPU, sizeElementX * sizeElementY * sizeType);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
}

void GPUController::InitializeLocal(int* errorStatus) {
    cudaStatus = hipHostMalloc(&pointer_Stream_CPU_input, sizeElementY * sizeElementZ * sizeType);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    cudaStatus = hipHostMalloc(&pointer_Stream_CPU_result, sizeElementX * sizeElementZ * sizeType);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    cudaStatus = hipMalloc(&pointer_Stream_GPU_input, sizeElementY * sizeElementZ * sizeType);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    cudaStatus = hipMalloc(&pointer_Stream_GPU_result, sizeElementX * sizeElementZ * sizeType);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
}

void GPUController::LoadGlobal(void* pointer, int* errorStatus) {
    unsigned size = sizeElementX * sizeElementY;
    if (sizeType == 8) {
        for (unsigned i = 0u; i < size; i++) {
            ((double*)pointer_CPU)[i] = ((double*)pointer)[i];
        }
    }
    else {
        *errorStatus = 1;
        return;
    }
    hipblasStatus_t = hipblasSetMatrixAsync(sizeElementX, sizeElementY, sizeType, pointer_CPU, sizeElementX, pointer_GPU, sizeElementX, cudaStream);
    RAISE_CUBLAS_ERROR(hipblasStatus_t, errorStatus);
}

void GPUController::LoadLocal(void* pointer, int* errorStatus) {
    unsigned size = sizeElementY * sizeElementZ;
    if (sizeType == 8) {
        for (unsigned i = 0u; i < size; i++) {
            ((double*)pointer_Stream_CPU_input)[i] = ((double*)pointer)[i];
        }
    }
    else {
        *errorStatus = 1;
        return;
    }
    hipblasStatus_t = hipblasSetMatrixAsync(sizeElementY, sizeElementZ, sizeType, pointer_Stream_CPU_input, sizeElementX, pointer_Stream_GPU_input, sizeElementY, cudaStream);
    RAISE_CUBLAS_ERROR(hipblasStatus_t, errorStatus);
}

void GPUController::LaunchTask(int* errorStatus) {
    hipblasStatus_t = hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        (int)sizeElementX, (int)sizeElementZ, (int)sizeElementY,
        alpha, (const double*)pointer_GPU, (int)sizeElementX, (const double*)pointer_Stream_GPU_input, (int)sizeElementY,
        beta, (double*)pointer_Stream_GPU_result, (int)sizeElementX);
    RAISE_CUBLAS_ERROR(hipblasStatus_t, errorStatus)
}

void GPUController::UnloadLocal(void* pointer, int* errorStatus) {
    unsigned size = sizeElementX * sizeElementZ;
    hipblasStatus_t = hipblasGetMatrixAsync(sizeElementX, sizeElementZ, sizeType, pointer_Stream_GPU_result, sizeElementX, pointer_Stream_CPU_result, sizeElementX, cudaStream);
    RAISE_CUBLAS_ERROR(hipblasStatus_t, errorStatus);
    cudaStatus = hipStreamSynchronize(cudaStream);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    if (sizeType == 8) {
        for (unsigned i = 0u; i < size; i++) {
            ((double*)pointer)[i] = ((double*)pointer_Stream_CPU_result)[i];
        }
    }
    else {
        *errorStatus = 1;
        return;
    }
}

void GPUController::FreeGlobal(int* errorStatus) {
    cudaStatus = hipFree(pointer_GPU);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    cudaStatus = hipHostFree(pointer_CPU);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
}

void GPUController::FreeLocal(int* errorStatus) {
    cudaStatus = hipFree(pointer_Stream_GPU_result);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    cudaStatus = hipFree(pointer_Stream_GPU_input);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    cudaStatus = hipHostFree(pointer_Stream_CPU_result);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
    cudaStatus = hipHostFree(pointer_Stream_CPU_input);
    RAISE_CUDA_ERROR(cudaStatus, errorStatus);
}

//Default Destructor for the GPU Controller class.
GPUController::~GPUController() {
    int errorStatus = 0;
    hipblasStatus_t = hipblasDestroy(cublasHandle);
    RAISE_CUBLAS_ERROR(hipblasStatus_t, &errorStatus);
    cudaStatus = hipStreamDestroy(cudaStream);
    RAISE_CUDA_ERROR(cudaStatus, &errorStatus);
    delete alpha;
    delete beta;
}
